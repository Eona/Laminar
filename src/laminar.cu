#include "hip/hip_runtime.h"
/*
 * Eona Studio (c)2015
 */
#include "connection.h"
#include "full_connection.h"
#include "gated_connection.h"
#include "loss_layer.h"
#include "activation_layer.h"
#include "bias_layer.h"
#include "parameter.h"
#include "network.h"
#include "lstm.h"
#include "rnn.h"
#include "learning_session.h"
#include "optimizer.h"
#include "gradient_check.h"

#include "engine/engine.h"
#include "engine/tensor.h"
#include "engine/tensor_ops.h"

#include "backend/dummy/dummy_engine.h"
#include "backend/dummy/dummy_dataman.h"
#include "backend/vecmat/vecmat_engine.h"
#include "backend/vecmat/vecmat_rand_dataman.h"
#include "backend/vecmat/vecmat_func_dataman.h"
#include "utils/global_utils.h"
#include "utils/timer.h"

#include "demo/mnist/mnist_parser.h"

FakeRand& rand_conn = FakeRand::instance_connection();
FakeRand& rand_prehis = FakeRand::instance_prehistory();
FakeRand& rand_input = FakeRand::instance_input();
FakeRand& rand_target = FakeRand::instance_target();

#define conn_full Connection::make<FullConnection>
#define conn_const Connection::make<ConstantConnection>
#define conn_gated Connection::make<GatedConnection>

template<typename EngineT>
struct PrintGradient : public Observer<Network>
{
	int maxEpoch;

	PrintGradient(int maxEpoch) :
		maxEpoch(maxEpoch)
	{}

	void observe(Network::Ptr net, LearningState::Ptr state)
	{
		if (false)
		if (state->batchInEpoch == 0 && state->epoch == maxEpoch - 1)
		{
			auto params = net->param_containers();
			DEBUG_TITLE("param gradient");
			for (int i = 0; i < params.size(); ++i)
				DEBUG_MSG(*net->get_engine<EngineT>()->read_memory(params[i]->param_gradient(0)));
			DEBUG_TITLE("param values");
			for (int i = 0; i < params.size(); ++i)
				DEBUG_MSG(*net->get_engine<EngineT>()->read_memory(params[i]->param_value(0)));
		}
	}
};

int main(int argc, char **argv)
{
	auto images = read_mnist_image("../data/t10k-images-idx3-ubyte", 100);
	auto labels = read_mnist_label("../data/t10k-labels-idx1-ubyte", 100);

	lmn::Vecmatf mat(28, 28);
	mat.fill([&](int r, int c) {
		return images[4][28 * r + c];
	});
	DEBUG_MSG(mat);
	mat.fill([&](int r, int c) {
		return images[7][28 * r + c];
	});
	DEBUG_MSG(mat);

	DEBUG_MSG(labels);

/*
//	const int HISTORY = 5;
	const int INPUT_DIM = 4;
	const int TARGET_DIM = 3;
	const int BATCH = 2;
	const int MAX_EPOCH = 1000;

	rand_conn.gen_uniform_rand(90, -0.1, 0.1, DEBUG_SEED); //rand_conn.print_rand_seq();
//
//	rand_prehis.gen_uniform_rand(30, -.5, .5); //rand_prehis.print_rand_seq();
//
//	rand_input.gen_uniform_rand(20, -1, 1); //rand_input.print_rand_seq();
//
//	rand_target.gen_uniform_rand(40, -1, 1); //rand_target.print_rand_seq();

	auto learnableFunc = [](const lmn::Vecmatf& in, lmn::Vecmatf& out) {
		// Each column is a batch
		for (int c = 0; c < in.col(); ++c)
		{
//			out(0, c) = sin(in(0, c)) + cos(in(1, c));
//			out(1, c) = cos(in(1, c)) + sin(in(2, c));
//			out(2, c) = 2 * sin(in(2, c)) - cos(in(3, c));
			out(0, c) = cos(in(0, c)) * in(1, c) + in(2, c);
			out(1, c) = in(1, c) * sin(in(2, c)) + in(3, c);
			out(2, c) = in(2, c) * in(3, c) + sin(in(0, c));
		}
	};

	auto engine = EngineBase::make<VecmatEngine>();
	auto dataman = DataManagerBase::make<VecmatFuncDataManager>(
						engine, INPUT_DIM, TARGET_DIM, BATCH,
						learnableFunc,
						100, 20, 10,
						-1.f, 1.f);

	auto linput = Layer::make<ConstantLayer>(INPUT_DIM);
	auto l2 = Layer::make<SigmoidLayer>(100);
	auto l3 = Layer::make<SigmoidLayer>(100);
	auto lloss = Layer::make<SquareLossLayer>(TARGET_DIM);

	auto net = ForwardNetwork::make(engine, dataman);
	net->add_layer(linput);
	net->new_connection<FullConnection>(linput, l2);
	net->new_bias_layer(l2);
	net->add_layer(l2);
	net->new_connection<FullConnection>(l2, l3);
	net->new_bias_layer(l3);
	net->add_layer(l3);
	net->new_connection<FullConnection>(l3, lloss);
	net->add_layer(lloss);

	auto opm = Optimizer::make<SGD>(0.3);
	auto eval = NoMetricEvaluator<VecmatEngine>::make(net);
	auto stopper = StopCriteria::make<MaxEpochStopper>(MAX_EPOCH);
	auto ser = NullSerializer::make();
	auto sched = EpochIntervalSchedule::make(1, 1);

	auto session = new_learning_session(net, opm, eval, stopper, ser, sched,
			std::make_shared<PrintGradient<VecmatEngine>>(MAX_EPOCH));

	session->initialize();

	session->train();
*/


	/*Tensor t1(dummyEng, { 2, 3 });
	Tensor t2(dummyEng, {5, 7});
	Tensor t3 = t1 + t2;
	Scalor s1(dummyEng);
	Scalor s2(dummyEng);

//	t1 -= t2;
//	t1 += t2;
//	t1 *= t2;
//	t1 *= s2;
//	s1 *= s2;
//	s1 += s2;
//	s1 -= s2;

	cout << "t3 " << t3.addr << endl;
	t3 = t3 + t3 - t1;
	cout << "t3 " << t3.addr << endl;
	t1 = t3 + 6.6f*t1 + t3;
	t3 = t1 * 3.5f;
	t1 *= 100.88f;
	cout << "t3 " << t3.addr << endl;

	dummyEng->print_routines();
	dummyEng->flush_execute();*/


/*	dummyEng->print_instructions();
	print_title("optimize");
	dummyEng->eliminate_temporary();
	dummyEng->print_instructions();

	for (auto pr : dummyEng->memoryPool.dimensions)
		DEBUG_MSG(pr.first << " " << pr.second);

	print_title("Graph");
	dummyEng->construct_graph();
	dummyEng->print_graph();*/

	DEBUG_TITLE("DONE");
}
