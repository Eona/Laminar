#include "hip/hip_runtime.h"
/*
 * Eona Studio (c)2015
 */
#include "connection.h"
#include "full_connection.h"
#include "gated_connection.h"
#include "loss_layer.h"
#include "activation_layer.h"
#include "bias_layer.h"
#include "parameter.h"
#include "network.h"
#include "lstm.h"
#include "rnn.h"
#include "learning_session.h"
#include "optimizer.h"
#include "gradient_check.h"

#include "engine/engine.h"
#include "engine/tensor.h"
#include "engine/tensor_ops.h"

#include "backend/dummy/dummy_engine.h"
#include "backend/dummy/dummy_dataman.h"
#include "backend/vecmat/vecmat_engine.h"
#include "backend/vecmat/vecmat_rand_dataman.h"
#include "backend/vecmat/vecmat_func_dataman.h"
#include "backend/cublas/cuda_engine.h"
#include "backend/types/cuda_float_mat.h"
#include "backend/opencl/opencl_engine.h"
#include "utils/global_utils.h"
#include "utils/timer.h"

#include "demo/mnist/mnist_parser.h"

FakeRand& rand_conn = FakeRand::instance_connection();
FakeRand& rand_prehis = FakeRand::instance_prehistory();
FakeRand& rand_input = FakeRand::instance_input();
FakeRand& rand_target = FakeRand::instance_target();

#define conn_full Connection::make<FullConnection>
#define conn_const Connection::make<ConstantConnection>
#define conn_gated Connection::make<GatedConnection>

template<typename EngineT>
struct PrintGradient : public Observer<Network>
{
	int maxEpoch;

	PrintGradient(int maxEpoch) :
		maxEpoch(maxEpoch)
	{}

	void observe(Network::Ptr net, LearningState::Ptr state)
	{
		if (false)
		if (state->batchInEpoch == 0 && state->epoch == maxEpoch - 1)
		{
			auto params = net->param_containers();
			DEBUG_TITLE("param gradient");
			for (int i = 0; i < params.size(); ++i)
				DEBUG_MSG(*net->get_engine<EngineT>()->read_memory(params[i]->param_gradient(0)));
			DEBUG_TITLE("param values");
			for (int i = 0; i < params.size(); ++i)
				DEBUG_MSG(*net->get_engine<EngineT>()->read_memory(params[i]->param_value(0)));
		}
	}
};

int main(int argc, char **argv)
{
	auto images = read_mnist_image(string("../data/mnist/") + MnistTrainImageFile, 10, 3, false);
	auto mnlabels = read_mnist_label(string("../data/mnist/") + MnistTrainLabelFile, 100);

	lmn::Vecmatf mat(28, 28);
	mat.fill([&](int r, int c) {
		return images[1][28 * r + c];
	});
	DEBUG_MSG(mat);
	mat.fill([&](int r, int c) {
		return images[1][2*28*28 + 28 * r + c];
	});
	DEBUG_MSG(mat);

	DEBUG_MSG(mnlabels);

	using namespace lmn::VecmatImpl;
	using lmn::Vecmatf;

	Vecmat<float> A = {
		{2.1, -1.2},
		{-3.3, .4},
		{1.65, -.7},
		{-1.33, .57}
	};

	Vecmat<float> A2 = {
			{1.1, 3, 6},
			{7.8, 2, 10},
			{5.9, 5, 5}
	};

	Vecmat<float> B = {
			{1, 2, 0}
	};

	auto a = std::make_shared<Vecmatf>(A2);
	auto ans = std::make_shared<Vecmatf>(A2.row(), A2.col());
	auto labels = std::make_shared<Vecmatf>(B);

	auto scalar = std::make_shared<Vecmatf>(1, 1);

	softmax({a}, ans, true);
	label_entropy_loss({ans, labels}, scalar, true);
	label_softmax_entropy_gradient({ans, labels}, ans, true);

	DEBUG_MSG(*scalar);
	DEBUG_MSG(*ans);

	/************************************/
	auto engine = std::make_shared<OpenclEngine>();
	Tensor t1(engine, {4, 7});
	Tensor t2(engine, {7, 9});

	lmn::fill_rand(t1);
	lmn::fill_rand(t2);
	engine->flush_execute();
	auto mem1 = engine->read_memory(t1);
	auto mem2 = engine->read_memory(t2);
	mem1->print_matrix("mem1");
	mem2->print_matrix("mem2");


	Tensor t3 = t1 * t2;

	engine->flush_execute();

	auto mem = engine->read_memory(t3);
	mem->print_matrix("fei shen");

/*
//	const int HISTORY = 5;
	const int INPUT_DIM = 4;
	const int TARGET_DIM = 3;
	const int BATCH = 2;
	const int MAX_EPOCH = 1000;

	rand_conn.gen_uniform_rand(90, -0.1, 0.1, DEBUG_SEED); //rand_conn.print_rand_seq();
//
//	rand_prehis.gen_uniform_rand(30, -.5, .5); //rand_prehis.print_rand_seq();
//
//	rand_input.gen_uniform_rand(20, -1, 1); //rand_input.print_rand_seq();
//
//	rand_target.gen_uniform_rand(40, -1, 1); //rand_target.print_rand_seq();

	auto learnableFunc = [](const lmn::Vecmatf& in, lmn::Vecmatf& out) {
		// Each column is a batch
		for (int c = 0; c < in.col(); ++c)
		{
//			out(0, c) = sin(in(0, c)) + cos(in(1, c));
//			out(1, c) = cos(in(1, c)) + sin(in(2, c));
//			out(2, c) = 2 * sin(in(2, c)) - cos(in(3, c));
			out(0, c) = cos(in(0, c)) * in(1, c) + in(2, c);
			out(1, c) = in(1, c) * sin(in(2, c)) + in(3, c);
			out(2, c) = in(2, c) * in(3, c) + sin(in(0, c));
		}
	};

	auto engine = EngineBase::make<VecmatEngine>();
	auto dataman = DataManagerBase::make<VecmatFuncDataManager>(
						engine, INPUT_DIM, TARGET_DIM, BATCH,
						learnableFunc,
						100, 20, 10,
						-1.f, 1.f);

	auto linput = Layer::make<ConstantLayer>(INPUT_DIM);
	auto l2 = Layer::make<SigmoidLayer>(100);
	auto l3 = Layer::make<SigmoidLayer>(100);
	auto lloss = Layer::make<SquareLossLayer>(TARGET_DIM);

	auto net = ForwardNetwork::make(engine, dataman);
	net->add_layer(linput);
	net->new_connection<FullConnection>(linput, l2);
	net->new_bias_layer(l2);
	net->add_layer(l2);
	net->new_connection<FullConnection>(l2, l3);
	net->new_bias_layer(l3);
	net->add_layer(l3);
	net->new_connection<FullConnection>(l3, lloss);
	net->add_layer(lloss);

	auto opm = Optimizer::make<SGD>(0.3);
	auto eval = NoMetricEvaluator<VecmatEngine>::make(net);
	auto stopper = StopCriteria::make<MaxEpochStopper>(MAX_EPOCH);
	auto ser = NullSerializer::make();
	auto sched = EpochIntervalSchedule::make(1, 1);

	auto session = new_learning_session(net, opm, eval, stopper, ser, sched,
			std::make_shared<PrintGradient<VecmatEngine>>(MAX_EPOCH));

	session->initialize();

	session->train();
*/


	/*Tensor t1(dummyEng, { 2, 3 });
	Tensor t2(dummyEng, {5, 7});
	Tensor t3 = t1 + t2;
	Scalar s1(dummyEng);
	Scalar s2(dummyEng);

//	t1 -= t2;
//	t1 += t2;
//	t1 *= t2;
//	t1 *= s2;
//	s1 *= s2;
//	s1 += s2;
//	s1 -= s2;

	cout << "t3 " << t3.addr << endl;
	t3 = t3 + t3 - t1;
	cout << "t3 " << t3.addr << endl;
	t1 = t3 + 6.6f*t1 + t3;
	t3 = t1 * 3.5f;
	t1 *= 100.88f;
	cout << "t3 " << t3.addr << endl;

	dummyEng->print_routines();
	dummyEng->flush_execute();*/


/*	dummyEng->print_instructions();
	print_title("optimize");
	dummyEng->eliminate_temporary();
	dummyEng->print_instructions();

	for (auto pr : dummyEng->memoryPool.dimensions)
		DEBUG_MSG(pr.first << " " << pr.second);

	print_title("Graph");
	dummyEng->construct_graph();
	dummyEng->print_graph();*/

	DEBUG_TITLE("DONE");
}
